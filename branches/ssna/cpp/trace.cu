#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "trace_cu.h"
#include "trace_utils.h"

#define USE_TEXLOOKUP

#define INIT_THREAD \
  const int xi = blockIdx.x * blockDim.x + threadIdx.x; \
  const int yi = blockIdx.y * blockDim.y + threadIdx.y; \
  const int sx = rp.viewWidth;                          \
  const int sy = rp.viewHeight;                         \
  if (xi >= sx || yi >= sy ) return; \
  const int tid = yi*sx + xi;        \

__constant__ VoxStructTree tree;
__constant__ RenderParams rp;


__constant__ float NoiseBuf[NoiseBufSize];
__constant__ float BlurZKernel[BlurZKernSize][BlurZKernSize];
const int halfKernSize = BlurZKernSize / 2;

__constant__ float EdgeThresholdCoef = 8.0f;

texture<uint, 1, hipReadModeElementType> nodes_tex;


#define GET_FIELD( id, fld ) ( tree.nodes[id].fld )

#ifdef USE_TEXLOOKUP
  #define NODE_SZ (sizeof(VoxNode)/4)
  #define GET_TEXNODE_FIELD( p, fld ) ( tex1Dfetch(nodes_tex, (p)+(fld)) )

  typedef uint NodePtr;
  __constant__ NodePtr InvalidPtr = 0xffffffff;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return id*NODE_SZ;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p/NODE_SZ; }

  __device__ VoxNodeInfo GetNodeInfo(NodePtr p) { return GET_TEXNODE_FIELD(p, 0); }
  __device__ VoxNodeId   GetParent  (NodePtr p) { return GET_TEXNODE_FIELD(p, 1); }
  __device__ VoxChild    GetChild   (NodePtr p, int chId) { return GET_TEXNODE_FIELD(p, 3 + chId); }
#else
  typedef const VoxNode * NodePtr;
  __constant__ NodePtr InvalidPtr = NULL;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return tree.nodes + id;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p - tree.nodes; }

  __device__ const VoxNodeInfo & GetNodeInfo(NodePtr p) { return p->flags; }
  __device__ const VoxNodeId   & GetParent  (NodePtr p) { return p->parent; }
  __device__ const VoxChild    & GetChild   (NodePtr p, int chId) { return p->child[chId]; }
#endif

__device__ VoxData GetVoxData  (VoxNodeId id) { return tree.nodes[id].data; }


__device__ float3 CalcRayDirView(int xi, int yi)
{
  const int sx = rp.viewWidth;
  const int sy = rp.viewHeight;
  return point_3f(2*rp.fovCoef*(float)(xi-sx/2)/sx, 2*rp.fovCoef*(float)(yi-sy/2)/sx, -1);
}

__device__ float3 CalcRayDirWorld(int xi, int yi)
{
  point_3f dir = CalcRayDirView(xi, yi);
  dir = rp.viewToWldMtx * dir;
  dir -= rp.eyePos;
  return dir;
}

/*__global__ void InitFishEyeRays(RayData * rays)
{
  INIT_THREAD

  
  float2 p = make_float2(xi-sx/2, yi-sy/2);
  p /= 0.5f*sx;

  const float pi = 3.141593f;
  p *= 0.8;

  float r = length(p);
  float phi = atan2(p.y, p.x);
  float theta = pi/2-asin(r);
  
  float ct = __cosf(theta);
  float3 v = make_float3(__cosf(phi)*ct, __sinf(phi)*ct, __sinf(theta));

  float3 dir = v.x*rp.right + v.y*rp.up + v.z*rp.dir;
  dir = normalize(dir);        

  rays[tid].dir.x = dir.x;
  rays[tid].dir.y = dir.y;
  rays[tid].dir.z = dir.z;

  rays[tid].endNode = 0;
  rays[tid].endNodeChild = EmptyNode;
}*/


const int MaxLeavLevel = 8;
__constant__ float LeafLevelProb[MaxLeavLevel] = {1.0, 0.8, 0.8, 0.7, 0.7, 0.7, 0.7, 0.7};

__device__ bool TraceLeaf(point_3f & t1, point_3f & t2, float nodeSize, const uint & dirFlags, int seed)
{
  return true;
  int childId = 0;
  int level = 0;
  int x(0), y(0), z(0);

  enum States { ST_EXIT, ST_ENTER, ST_GOUP, ST_GODOWN, ST_GONEXT };
  int state = ST_ENTER;
  while (state != ST_EXIT)
  {
    switch (state)
    {
      case ST_ENTER:
      {
        // t1, t2 -- for cur node
        int key = (x*11+y*23+z*57 + seed) % NoiseBufSize;
        if (NoiseBuf[key] > LeafLevelProb[level]) { state = ST_GOUP; break; }
        
        if (level == MaxLeavLevel-1 || maxCoord(t1) * rp.detailCoef > nodeSize) 
          return true;

        childId = FindFirstChild(t1, t2);
        state = ST_GODOWN; 
        break;
      }

      case ST_GODOWN:
      {
        if (minCoord(t2) < 0) { state = ST_GONEXT; break; }

        // t1, t2 -- for childId
        int wldCh = childId ^ dirFlags;
        x = x<<1 | (wldCh & 1 );
        y = y<<1 | ((wldCh>>1) & 1);
        z = z<<1 | ((wldCh>>2) & 1);
        ++level;
        nodeSize /= 2;

        state = ST_ENTER; 
        break;
      }

      case ST_GOUP:
      {
        // t1, t2 -- for cur node

        if (level == 0) { state = ST_EXIT; break; }

        childId = ((z&1)<<2 | (y&1)<<1 | (x&1)) ^ dirFlags;
        x >>= 1;
        y >>= 1;
        z >>= 1;
        --level;
        nodeSize *= 2;

        state = ST_GONEXT;
        break;
      }

      case ST_GONEXT:
      {
        // t1, t2 -- for childId
        if (GoNext(childId, t1, t2))
        {
          state = ST_GODOWN;
          break;
        }
        GoUp(childId, t1, t2);
        state = ST_GOUP;
        break;
      }
    }
  }
  return false;
}

__global__ void Trace()
{
  INIT_THREAD

  rp.rays[tid].endNode = 0;
  rp.rays[tid].endNodeChild = EmptyNode;
  rp.zbuf[tid] = 100.0;


  if (IsNull(rp.rays[tid].endNode))
    return;

  point_3f dir = CalcRayDirWorld(xi, yi);
  AdjustDir(dir);

  point_3f t1, t2;
  uint dirFlags = 0;
  if (!SetupTrace(rp.eyePos, dir, t1, t2, dirFlags))
  {
    rp.rays[tid].endNode = EmptyNode;
    return;
  }

  NodePtr nodePtr = GetNodePtr(tree.root);
  int childId = 0;
  float nodeSize = 1.0f;

  enum States { ST_EXIT, ST_ANALYSE, ST_SAVE, ST_GOUP, ST_GODOWN, ST_GONEXT };
  int state = ST_ANALYSE;
  while (state != ST_EXIT)
  {
    switch (state)
    {
      case ST_ANALYSE:
      {
        childId = -1;
        if (maxCoord(t1) * rp.detailCoef > nodeSize/2)  { state = GetEmptyFlag(GetNodeInfo(nodePtr)) ? ST_GOUP : ST_SAVE; break; }
        
        childId = FindFirstChild(t1, t2);
        state = ST_GODOWN;
        break;
      }
      
      case ST_GODOWN:
      {
        if (minCoord(t2) < 0) { state = ST_GONEXT; break; }

        if (GetLeafFlag(GetNodeInfo(nodePtr), childId^dirFlags)) 
        { 
          int seed = Ptr2Id(nodePtr) + 119*(childId^dirFlags);
          state = TraceLeaf(t1, t2, nodeSize / 2, dirFlags, seed) ? ST_SAVE : ST_GONEXT;
          break; 
        }
        
        VoxNodeId ch = GetChild(nodePtr, childId^dirFlags);
        if (IsNull(ch)) {state = ST_GONEXT; break; }
        nodePtr = GetNodePtr(ch);
        nodeSize /= 2;
        state = ST_ANALYSE;
        break;
      }
      
      case ST_GONEXT:
      {
        state = GoNext(childId, t1, t2) ? ST_GODOWN : ST_GOUP;
        break;
      }

      case ST_GOUP:
      {
        VoxNodeId p = GetParent(nodePtr);
        if (IsNull(p)) 
        { 
          rp.rays[tid].endNode = EmptyNode;
          state = ST_EXIT; 
          break; 
        }

        GoUp(childId, t1, t2);
        childId = GetSelfChildId(GetNodeInfo(nodePtr))^dirFlags;
        nodePtr = GetNodePtr(p);
        nodeSize *= 2;
        state = ST_GONEXT;
        break;
      }

      case ST_SAVE:
      {
        rp.rays[tid].endNode = Ptr2Id(nodePtr);
        rp.rays[tid].endNodeChild = childId^dirFlags;
        rp.zbuf[tid] = maxCoord(t1);
        rp.rays[tid].endNodeSize = nodeSize;
        state = ST_EXIT;
        break;
      }
    }
  }
}

__device__ point_3f CalcLighting(point_3f pos, point_3f normal, point_3f color)
{
  point_3f accum = rp.ambient * color;
  for (int i = 0; i < MaxLightsNum; ++i)
  {
    if (!rp.lights[i].enabled)
      continue;

    point_3f ligthPos = rp.wldToViewMtx * rp.lights[i].pos;
    point_3f lightDir = ligthPos - pos;
    float lightDist2 = dot(lightDir, lightDir);
    float lightDist = sqrtf(lightDist2);
    float attenuation = 1.0f / dot(point_3f(1.0f, lightDist, lightDist2), rp.lights[i].attenuationCoefs);
    lightDir /= lightDist;

    point_3f diffuse = rp.lights[i].diffuse * color * max(dot(lightDir, normal), 0.0f);
    
    point_3f viewerDir = normalize(point_3f(0, 0, 0) - pos);
    point_3f hv = normalize(viewerDir + lightDir);
    point_3f specular = rp.lights[i].specular * pow(max(0.0f, dot(hv, normal)), rp.specularExp);

    accum += (diffuse + specular) * attenuation;
  }
  return accum;
}

__device__ float absmin(float a, float b)
{
  return abs(a) <  abs(b) ? a : b;
}

__device__ float approxDeriv(float z0, float z1, float z2)
{
  float threshold = EdgeThresholdCoef * z1 * rp.pixelAng;

  float dz = 0;
  float c = 0;
  float d1 = z1 - z0;
  float d2 = z2 - z1;
  if (abs(d1) < threshold)
  {
    dz += d1;
    c += 1.0f;
  }
  if (abs(d2) < threshold)
  {
    dz += d2;
    c += 1.0f;
  }
  if (c == 0)
    return (d1 + d2) / 2;
  return dz / c;
}

__device__ point_3f SampleNormal(int xi, int yi, const float * zbuf)
{
  int tid = yi * rp.viewWidth + xi;
  if (xi == 0 || yi == 0 || xi == rp.viewWidth-1 || yi == rp.viewHeight-1 )
    return point_3f(0, 0, 1);
  
  float z = zbuf[tid];
  float zl = zbuf[tid-1];
  float zr = zbuf[tid+1];
  float zd = zbuf[tid-rp.viewWidth];
  float zu = zbuf[tid+rp.viewWidth];

  float dx = approxDeriv(zl, z, zr);
  float dy = approxDeriv(zd, z, zu);
  
  //nx = -d * dx * z
  //ny = -d * dy * z
  //nz = d * d * z * z
  float d = 2*rp.fovCoef / rp.viewWidth;
  point_3f n(dx, dy, d*z);
  n *= d*z;
  return normalize(n);
}

__global__ void ShadeSimple(uchar4 * img, const float * zbuf )
{
  INIT_THREAD;

  VoxNodeId node = rp.rays[tid].endNode;
  if (IsNull(node))
  {
    img[tid] = make_uchar4(0, node == EmptyNode ? 0 : 64, 0, 255);
    return;
  }

  float3 dir = CalcRayDirView(xi, yi);
  float dl = length(dir);
  dir /= dl;
  float t = zbuf[tid] / dl;

  VoxData vd;
  int childId = rp.rays[tid].endNodeChild;
  if (childId < 0)
    vd = GetVoxData(node);
  else
    vd = GetChild(GetNodePtr(node), childId);

  Color16  c16;
  Normal16 n16;
  UnpackVoxData(vd, c16, n16);
  uchar4 col;
  col = UnpackColorCU(c16);

  point_3f norm;
  if (!rp.ssna) // ((xi/64 + yi/64) & 1) == 0
  {
    UnpackNormal(n16, norm.x, norm.y, norm.z);
    norm = rp.wldToViewMtx * (norm + rp.eyePos);
  }
  else
    norm = SampleNormal(xi, yi, zbuf);


  float3 pt = dir*t;
  point_3f materialColor = point_3f(col.x, col.y, col.z) / 256.0f;
  point_3f res = fminf(CalcLighting(pt, norm, materialColor) * 256.0f, point_3f(255, 255, 255));

  if (rp.showNormals)
    res = norm*255;

  img[tid] = make_uchar4(res.x, res.y, res.z, 255);
}

__global__ void BlurZ(float farLimit, const float * src, float * dst)
{
  INIT_THREAD;
  
  int x1 = max(xi - halfKernSize, 0);
  int x2 = min(xi + halfKernSize, rp.viewWidth-1);
  int y1 = max(yi - halfKernSize, 0);
  int y2 = min(yi + halfKernSize, rp.viewHeight-1);

  int kx = xi - halfKernSize;
  int ky = yi - halfKernSize;

  float z = src[tid];
  if (z > farLimit)
  {
    dst[tid] = z;
    return;
  }
  float threshold = EdgeThresholdCoef * z * rp.pixelAng;
  threshold = max(threshold, EdgeThresholdCoef / 2048);


  float acc = 0, count = 0;
  for (int y = y1; y <= y2; ++y)
  {
    for (int x = x1; x <= x2; ++x)
    {
      float s = src[y * rp.viewWidth + x];
      float dz = s - z;
      if (dz > threshold) // too far
      {
        s = z + threshold / EdgeThresholdCoef;
      }
      else if (dz < - threshold) // too close
        s = z;

      float k = BlurZKernel[y - ky][x - kx];
      acc += s * k;
      count += k;
    }
  }
  dst[tid] = acc / count;
}


extern "C" {

void Run_Trace(GridShape grid)
{
  Trace<<<grid.grid, grid.block>>>();
}

void Run_ShadeSimple(GridShape grid, uchar4 * img, const float * zbuf)
{
  ShadeSimple<<<grid.grid, grid.block>>>(img, zbuf);
}

void Run_BlurZ(GridShape grid, float farLimit, const float * src, float * dst)
{
  BlurZ<<<grid.grid, grid.block>>>(farLimit, src, dst);
}

}