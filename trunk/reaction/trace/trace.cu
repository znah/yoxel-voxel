#include "hip/hip_runtime.h"
//
#include "utils.h"

__constant__ float4x4 c_proj2wldMtx;
__constant__ float3   c_eyePos;
__constant__ int2     c_viewSize;

texture<uchar,  3, hipReadModeNormalizedFloat> volumeTex;   // 3D texture
texture<float4, 1, hipReadModeElementType>     transferTex; // 1D transfer function texture
texture<uchar,  3, hipReadModeElementType>     markTex;

struct RayData
{
    float3 orig;
    float3 dir;
};

__device__ RayData PrepareRay(int x, int y)
{
    const float eps = 1e-5f;
    float px = 2.0f * (float)x / c_viewSize.x - 1.0f;
    float py = 2.0f * (float)y / c_viewSize.y - 1.0f;
    float4 v = make_float4( px, py, 0, 1 );
    v = mul(c_proj2wldMtx, v);
    float invW = 1.0f / v.w;
    v.x *= invW;
    v.y *= invW;
    v.z *= invW;
    float3 dir = normalize(make_float3(v) - c_eyePos);
    if (fabsf(dir.x) < eps) dir.x = copysignf(eps, dir.x);
    if (fabsf(dir.y) < eps) dir.y = copysignf(eps, dir.y);
    if (fabsf(dir.z) < eps) dir.z = copysignf(eps, dir.z);
    
    RayData ray;
    ray.orig = c_eyePos;
    ray.dir  = dir;
    return ray;
}


__device__ bool walkBrick(RayData ray, float t_enter, float t_exit, float4 & accum)
{
    const float dt = 1.0f / 1024.0f;
    for (float t = t_enter; t < t_exit; t += dt)
    {
        float3 p = ray.orig + t * ray.dir;
        float v = tex3D(volumeTex, p.x, p.y, p.z);
        float4 col = make_float4(saturate( (v-0.5f)*4.0 ));
        col.w *= 0.5;
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;

        accum += col * (1.0 - accum.w);
        if (accum.w > 0.99f)
            return true;
    }
    return false;
}


__device__ float3 intersect(float3 orig, float3 dt, float3 lo, float3 hi, float & t_enter, float & t_exit)
{
    float3 tlo = dt*(lo - orig);
    float3 thi = dt*(hi - orig);

    float3 t1 = fminf(tlo, thi);
    float3 t2 = fmaxf(tlo, thi);

    t_enter = max(t1.x, max(t1.y, t1.z));
    t_exit  = min(t2.x, min(t2.y, t2.z));

    if (t_exit == t2.x) return make_float3(1, 0, 0);
    if (t_exit == t2.y) return make_float3(0, 1, 0);
    return make_float3(0, 0, 1);
}


__device__ float4 castRay(RayData ray)
{
    /*float3 t_coef, t_bias;
    t_coef.x = 1.0f / fabs(ray.dir.x);
    t_coef.y = 1.0f / fabs(ray.dir.y);
    t_coef.z = 1.0f / fabs(ray.dir.z);

    t_bias.x = -t_coef.x * ray.orig.x;
    t_bias.y = -t_coef.y * ray.orig.y;
    t_bias.z = -t_coef.z * ray.orig.z;

    int octant_mask = 0;
    if (ray.dir.x < 0.0f) octant_mask ^= 1, t_bias.x = -t_coef.x - t_bias.x;
    if (ray.dir.y < 0.0f) octant_mask ^= 2, t_bias.y = -t_coef.y - t_bias.y;
    if (ray.dir.z < 0.0f) octant_mask ^= 4, t_bias.z = -t_coef.z - t_bias.z;*/

    float3 dt = make_float3(1.0f) / ray.dir;

    float t_enter, t_exit;
    intersect(ray.orig, dt, make_float3(0.0), make_float3(1.0), t_enter, t_exit);

    if (t_exit < 0.0f || t_enter > t_exit)
        return make_float4(0, 0, 0, 0);

    const float gridSize  = 64.0f;
    const float brickSize = 1.0f / gridSize;

    float3 p = ray.orig + t_enter * ray.dir;
    p = make_float3(make_int3(p * gridSize)) * brickSize;

    float3 dp;
    dp.x = copysignf(1.0f, ray.dir.x);
    dp.y = copysignf(1.0f, ray.dir.y);
    dp.z = copysignf(1.0f, ray.dir.z);

    float t_stop = t_exit;

    float4 accum = make_float4(0);
    while (true)
    {
      float3 step = intersect(ray.orig, dt, p, p+brickSize, t_enter, t_exit) * dp;
      if (t_enter > t_stop)
        break;

      float3 p2 = p + brickSize*0.5f;
      //if (tex3D(markTex, p2.x, p2.y, p2.z) > 0)
        //walkBrick(ray, t_enter, t_exit, accum);
      accum.x += 0.1;
      p += step;
    }

    walkBrick(ray, t_enter, t_exit, accum);
    return accum;
}
 
extern "C"
__global__ void Trace(uint * d_img)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    RayData ray = PrepareRay(x, y);
    float4 res = castRay(ray);

    int ofs = x + y * c_viewSize.x;
    d_img[ofs] = float2uint(res);
}
