#include "hip/hip_runtime.h"
//
#include "utils.h"

__constant__ float4x4 c_proj2wldMtx;
__constant__ float3   c_eyePos;
__constant__ int2     c_viewSize;

texture<uchar,  3, hipReadModeNormalizedFloat> volumeTex;   // 3D texture
texture<float4, 1, hipReadModeElementType>     transferTex; // 1D transfer function texture


__device__ float3 calcRayDir(int x, int y)
{
    const float eps = 1e-5f;
    float px = 2.0f * (float)x / c_viewSize.x - 1.0f;
    float py = 2.0f * (float)y / c_viewSize.y - 1.0f;
    float4 v = make_float4( px, py, 0, 1 );
    v = mul(c_proj2wldMtx, v);
    float invW = 1.0f / v.w;
    v.x *= invW;
    v.y *= invW;
    v.z *= invW;
    float3 dir = normalize(make_float3(v) - c_eyePos);
    if (fabsf(dir.x) < eps) dir.x = copysignf(eps, dir.x);
    if (fabsf(dir.y) < eps) dir.y = copysignf(eps, dir.y);
    if (fabsf(dir.z) < eps) dir.z = copysignf(eps, dir.z);
    return dir;
}

__device__ float4 castRay(float3 orig, float3 dir)
{
    float tx_coef = 1.0f / fabs(dir.x);
    float ty_coef = 1.0f / fabs(dir.y);
    float tz_coef = 1.0f / fabs(dir.z);

    float tx_bias = -tx_coef * orig.x;
    float ty_bias = -ty_coef * orig.y;
    float tz_bias = -tz_coef * orig.z;

    int octant_mask = 0;
    if (dir.x < 0.0f) octant_mask ^= 1, tx_bias = -tx_coef - tx_bias;
    if (dir.y < 0.0f) octant_mask ^= 2, ty_bias = -ty_coef - ty_bias;
    if (dir.z < 0.0f) octant_mask ^= 4, tz_bias = -tz_coef - tz_bias;

    float t_enter = fmaxf(tx_bias, ty_bias, tz_bias);
    float t_exit  = fminf(tx_coef + tx_bias, ty_coef + ty_bias, tz_coef + tz_bias);
    
    t_enter = fmaxf(0.0f, t_enter);
    if (t_exit < 0.0f || t_enter > t_exit)
        return make_float4(0, 0, 0, 0);







    return make_float4(t_enter, t_exit, 0, 0);

}
 
extern "C"
__global__ void Trace(uint * d_img)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    float3 dir = calcRayDir(x, y);
    float4 res = castRay(c_eyePos, dir);

    int ofs = x + y * c_viewSize.x;
    d_img[ofs] = float2uint(res);
}
