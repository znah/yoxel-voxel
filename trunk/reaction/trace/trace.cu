#include "hip/hip_runtime.h"
//
#include "utils.h"

__constant__ float4x4 c_proj2wldMtx;
__constant__ float3   c_eyePos;
__constant__ int2     c_viewSize;

texture<uchar,  3, hipReadModeNormalizedFloat> volumeTex;   // 3D texture
texture<float4, 1, hipReadModeElementType>     transferTex; // 1D transfer function texture
texture<uchar,  3, hipReadModeElementType>     markTex;

struct RayData
{
    float3 orig;
    float3 dir;
};

__device__ RayData PrepareRay(int x, int y)
{
    const float eps = 1e-5f;
    float px = 2.0f * (float)x / c_viewSize.x - 1.0f;
    float py = 2.0f * (float)y / c_viewSize.y - 1.0f;
    float4 v = make_float4( px, py, 0, 1 );
    v = mul(c_proj2wldMtx, v);
    float invW = 1.0f / v.w;
    v.x *= invW;
    v.y *= invW;
    v.z *= invW;
    float3 dir = normalize(make_float3(v) - c_eyePos);
    if (fabsf(dir.x) < eps) dir.x = copysignf(eps, dir.x);
    if (fabsf(dir.y) < eps) dir.y = copysignf(eps, dir.y);
    if (fabsf(dir.z) < eps) dir.z = copysignf(eps, dir.z);
    
    RayData ray;
    ray.orig = c_eyePos;
    ray.dir  = dir;
    return ray;
}


__device__ bool walkBrick(RayData ray, float t_enter, float t_exit, float4 & accum)
{
    const float dt = 1.0f / 1024.0f;
    for (float t = t_enter; t < t_exit; t += dt)
    {
        float3 p = ray.orig + t * ray.dir;
        float v = tex3D(volumeTex, p.x, p.y, p.z);
        float4 col = make_float4(saturate( (v-0.5f)*4.0 ));
        col.w *= 0.5;
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;

        accum += col * (1.0 - accum.w);
        if (accum.w > 0.99f)
            return true;
    }
    return false;
}

__device__ float4 castRay(RayData ray)
{
    float3 t_coef, t_bias;
    t_coef.x = 1.0f / fabs(ray.dir.x);
    t_coef.y = 1.0f / fabs(ray.dir.y);
    t_coef.z = 1.0f / fabs(ray.dir.z);

    t_bias.x = -t_coef.x * ray.orig.x;
    t_bias.y = -t_coef.y * ray.orig.y;
    t_bias.z = -t_coef.z * ray.orig.z;

    int octant_mask = 0;
    if (ray.dir.x < 0.0f) octant_mask ^= 1, t_bias.x = -t_coef.x - t_bias.x;
    if (ray.dir.y < 0.0f) octant_mask ^= 2, t_bias.y = -t_coef.y - t_bias.y;
    if (ray.dir.z < 0.0f) octant_mask ^= 4, t_bias.z = -t_coef.z - t_bias.z;

    float t_enter = fmaxf(t_bias.x, t_bias.y, t_bias.z);
    float t_exit  = fminf(t_coef.x + t_bias.x, t_coef.y + t_bias.y, t_coef.z + t_bias.z);
    
    t_enter = fmaxf(0.0f, t_enter);
    if (t_exit < 0.0f || t_enter > t_exit)
        return make_float4(0, 0, 0, 0);


    const float gridSize  = 64.0f;
    const float brickSize = 1.0f / gridSize;

    float3 p = ray.orig + t_enter * ray.dir;
    p = make_float3(make_int3(p * gridSize)) * brickSize;

    float4 accum = make_float4(0);
    while (true)   
    {
      float3 t = (p + brickSize) * t_coef + t_bias;
      float t2 = fmaxf(t.x, t.y, t.z);
      if (t2 == t.x) p.x += brickSize;
      if (t2 == t.y) p.y += brickSize;
      if (t2 == t.z) p.z += brickSize;
      
      if (tex3D(markTex, p.x, p.y, p.z) != 0)
        walkBrick(ray, t_enter, t2, accum);
      t_enter = t2;
      if (t_enter >= t_exit)
        break;
    }
    return accum;
}
 
extern "C"
__global__ void Trace(uint * d_img)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    RayData ray = PrepareRay(x, y);
    float4 res = castRay(ray);

    int ofs = x + y * c_viewSize.x;
    d_img[ofs] = float2uint(res);
}
