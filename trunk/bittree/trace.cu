#include "hip/hip_runtime.h"
//
#include "cutil_math.h"
#include "cu_matrix.h"

// 31 bit 
// - grid(0)
// - brick(1)
// special:
// 0xFF FF FF F0 - all zero
// 0xFF FF FF F1 - all one

typedef uint node_id;

const node_id ZeroBlock    = 0xFFFFFFF0;
const node_id FullBlock    = 0xFFFFFFF1;
const node_id BrickRefMask = 0x80000000;

const int BrickSize = 4;
const int GridSize = 4;

const int GridSize3 = GridSize * GridSize * GridSize;


#pragma pack(push, 4)
struct RenderParams
{
  node_id hintTreeRoot;
  uint2 viewSize;

  float fovCoef; // tan(fov/2)

  float3 eyePos;
  float3x4 viewToWldMtx;
  float3x4 wldToViewMtx;
};
#pragma pack(pop)

__constant__ RenderParams rp;

texture<uint, 1, hipReadModeElementType> hint_grid_tex;
texture<uint2, 1, hipReadModeElementType> hint_brick_tex;




__device__ uint fetchHint(float3 pos)
{
  float3 nodePos = make_float3(0, 0, 0);
  node_id node = rp.hintTreeRoot;
  while ( (node & BrickRefMask) == 0 )
  {
    pos *= GridSize;
    float3 childPos = floor(pos);
    pos = pos - childPos;
    int childId = (childPos.z * GridSize + childPos.y) * GridSize + childPos.x;
    node = tex1Dfetch(hint_grid_tex, node * GridSize3 + childId);
  }
  if (node == ZeroBlock)
    return 0;
  else if (node == FullBlock)
    return 1;
  
  node &= ~BrickRefMask;
  pos *= BrickSize;
  float3 voxPos = floor(pos);
  int voxId = (voxPos.z * BrickSize + voxPos.y) * BrickSize + voxPos.x;
  uint2 brick = tex1Dfetch(hint_brick_tex, node);
  uint bits = (voxId < 32) ? brick.x : brick.y;
  return (bits >> (voxId & 0x1f)) & 1;
}


__device__ float3 CalcRayViewDir(int xi, int yi)
{
  const int sx = rp.viewSize.x;
  const int sy = rp.viewSize.y;
  float dl = 2.0f * rp.fovCoef / sx;
  return make_float3( dl*(xi-sx/2), dl*(yi-sy/2), -1 );
}

__device__ float3 CalcRayWorldDir(int xi, int yi)
{
  float3 dir = CalcRayViewDir(xi, yi);
  dir = mul(rp.viewToWldMtx, dir);
  return dir;
}
__device__ int sign(float v)
{
  return v > 0 ? 1 : (v < 0 ? -1 : 0);
}

__device__ bool hitBox(float3 dir, float3 orig, float3 boxMin, float3 boxMax, float3 & t1, float3 & t2)
{
  float3 invDir = make_float3(1.0f) / dir;
  float3 tt1 = invDir * (boxMin - orig);
  float3 tt2 = invDir * (boxMax - orig);
  t1 = fminf(tt1, tt2);
  t2 = fmaxf(tt1, tt2);
  float tenter = fmaxf( fmaxf(t1.x, t1.y), t1.z );
  float texit  = fminf( fminf(t2.x, t2.y), t2.z );

  return (texit > 0.0f) && (tenter < texit);
}






#define INIT_THREAD \
  const int xi = blockIdx.x * blockDim.x + threadIdx.x; \
  const int yi = blockIdx.y * blockDim.y + threadIdx.y; \
  if (xi >= rp.viewSize.x || yi >= rp.viewSize.y ) return; \
  const int tid = yi*rp.viewSize.x + xi;        \


extern "C" {

__global__ void TestFetch(float slice, float * dst)
{
  INIT_THREAD;
  
  float3 p = make_float3((float)xi / rp.viewSize.x, (float)yi / rp.viewSize.y, slice);
  dst[tid] = fetchHint(p);
}

__global__ void Trace(float * dst)
{
  INIT_THREAD;
  float3 dir = CalcRayWorldDir(xi, yi);
  int3 dirBits = make_int3( signbit(dir.x), signbit(dir.y), signbit(dir.z) );
  float3 t1, t2;
  if (!hitBox(dir, rp.eyePos, make_float3(0.0f), make_float3(1.0f), t1, t2))
  {
    dst[tid] = 0.0f;
    return;
  }






  float dzx = abs(dir.z / dir.x);
  float dyx = abs(dir.y / dir.x);
  float dzy = abs(dir.z / dir.y);
  float ezx = t2.x * abs(dir.z) + rp.eyePos.z;
  float eyx = t2.x * abs(dir.y) + rp.eyePos.y;
  float ezy = t2.y * abs(dir.z) + rp.eyePos.z;
                     

  dst[tid] = ezx;
  
}
 
}


