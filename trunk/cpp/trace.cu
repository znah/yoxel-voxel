#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "trace_cu.h"
#include "trace_utils.h"

#define USE_TEXLOOKUP

#define INIT_THREAD \
  const int xi = blockIdx.x * blockDim.x + threadIdx.x; \
  const int yi = blockIdx.y * blockDim.y + threadIdx.y; \
  const int sx = rp.viewWidth;                          \
  const int sy = rp.viewHeight;                         \
  if (xi >= sx || yi >= sy ) return; \
  const int tid = yi*sx + xi;        \

__constant__ VoxStructTree tree;
__constant__ RenderParams rp;

texture<uint, 1, hipReadModeElementType> nodes_tex;


#define GET_FIELD( id, fld ) ( tree.nodes[id].fld )

#ifdef USE_TEXLOOKUP
  #define NODE_SZ (sizeof(VoxNode)/4)
  #define GET_TEXNODE_FIELD( p, fld ) ( tex1Dfetch(nodes_tex, (p)+(fld)) )

  typedef uint NodePtr;
  __constant__ NodePtr InvalidPtr = 0xffffffff;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return id*NODE_SZ;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p/NODE_SZ; }

  __device__ VoxNodeInfo GetNodeInfo(NodePtr p) { return GET_TEXNODE_FIELD(p, 0); }
  __device__ VoxNodeId   GetParent  (NodePtr p) { return GET_TEXNODE_FIELD(p, 1); }
  __device__ VoxChild    GetChild   (NodePtr p, int chId) { return GET_TEXNODE_FIELD(p, 3 + chId); }
#else
  typedef const VoxNode * NodePtr;
  __constant__ NodePtr InvalidPtr = NULL;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return tree.nodes + id;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p - tree.nodes; }

  __device__ const VoxNodeInfo & GetNodeInfo(NodePtr p) { return p->flags; }
  __device__ const VoxNodeId   & GetParent  (NodePtr p) { return p->parent; }
  __device__ const VoxChild    & GetChild   (NodePtr p, int chId) { return p->child[chId]; }
#endif

__device__ VoxData GetVoxData  (VoxNodeId id) { return tree.nodes[id].data; }


__global__ void InitEyeRays(float * noiseBuf)
{
  INIT_THREAD

  float3 dir = rp.dir + 2*(xi-sx/2)*rp.right/sx + 2*(yi-sy/2)*rp.up/sy;
  dir = normalize(dir);

  int noiseBase = (tid*3 + rp.rndSeed) % (3*sx*sy-3);
  point_3f noiseShift = point_3f(noiseBuf[noiseBase], noiseBuf[noiseBase+1], noiseBuf[noiseBase+2]) * rp.ditherCoef;
  rp.rays[tid].pos = rp.eyePos + noiseShift;
  rp.rays[tid].dir = dir;

  rp.rays[tid].endNode = 0;
  rp.rays[tid].endNodeChild = EmptyNode;
}


/*__global__ void InitFishEyeRays(RayData * rays)
{
  INIT_THREAD

  
  float2 p = make_float2(xi-sx/2, yi-sy/2);
  p /= 0.5f*sx;

  const float pi = 3.141593f;
  p *= 0.8;

  float r = length(p);
  float phi = atan2(p.y, p.x);
  float theta = pi/2-asin(r);
  
  float ct = __cosf(theta);
  float3 v = make_float3(__cosf(phi)*ct, __sinf(phi)*ct, __sinf(theta));

  float3 dir = v.x*rp.right + v.y*rp.up + v.z*rp.dir;
  dir = normalize(dir);        

  rays[tid].dir.x = dir.x;
  rays[tid].dir.y = dir.y;
  rays[tid].dir.z = dir.z;

  rays[tid].endNode = 0;
  rays[tid].endNodeChild = EmptyNode;
}*/

__global__ void Trace()
{
  INIT_THREAD

  if (IsNull(rp.rays[tid].endNode))
    return;

  point_3f dir = rp.rays[tid].dir;
  AdjustDir(dir);

  point_3f t1, t2;
  uint dirFlags = 0;
  if (!SetupTrace(rp.rays[tid].pos, dir, t1, t2, dirFlags)) //rp.eyePos
  {
    rp.rays[tid].endNode = EmptyNode;
    return;
  }

  NodePtr nodePtr = GetNodePtr(tree.root);
  int childId = 0;
  int level = 0;
  float nodeSize = pow(0.5f, level);

  enum States { ST_EXIT, ST_ANALYSE, ST_SAVE, ST_GOUP, ST_GODOWN, ST_GONEXT };
  int state = ST_ANALYSE;
  while (state != ST_EXIT)
  {
    switch (state)
    {
      case ST_ANALYSE:
      {
        childId = -1;
        if (maxCoord(t1) * rp.detailCoef > nodeSize/2)  { state = GetEmptyFlag(GetNodeInfo(nodePtr)) ? ST_GOUP : ST_SAVE; break; }
        
        childId = FindFirstChild(t1, t2);
        state = ST_GODOWN;
        break;
      }
      
      case ST_GODOWN:
      {
        if (minCoord(t2) < 0) { state = ST_GONEXT; break; }

        if (GetLeafFlag(GetNodeInfo(nodePtr), childId^dirFlags)) { state = ST_SAVE; break; }
        
        VoxNodeId ch = GetChild(nodePtr, childId^dirFlags);
        if (IsNull(ch)) {state = ST_GONEXT; break; }
        nodePtr = GetNodePtr(ch);
        ++level;
        nodeSize /= 2;
        state = ST_ANALYSE;
        break;
      }
      
      case ST_GONEXT:
      {
        state = GoNext(childId, t1, t2) ? ST_GODOWN : ST_GOUP;
        break;
      }

      case ST_GOUP:
      {
        VoxNodeId p = GetParent(nodePtr);
        if (IsNull(p)) 
        { 
          rp.rays[tid].endNode = EmptyNode;
          state = ST_EXIT; 
          break; 
        }

        for (int i = 0; i < 3; ++i)
        {
          int mask = 1<<i;
          float dt = t2[i] - t1[i];
          ((childId & mask) == 0) ? t2[i] += dt : t1[i] -= dt;
        }
        childId = GetSelfChildId(GetNodeInfo(nodePtr))^dirFlags;
        nodePtr = GetNodePtr(p);
        --level;
        nodeSize *= 2;
        state = ST_GONEXT;
        break;
      }

      case ST_SAVE:
      {
        rp.rays[tid].endNode = Ptr2Id(nodePtr);
        rp.rays[tid].endNodeChild = childId^dirFlags;
        rp.rays[tid].t = maxCoord(t1);
        rp.rays[tid].endNodeSize = nodeSize;
        state = ST_EXIT;
        break;
      }
    }
  }
}

__device__ point_3f CalcLighting(point_3f pos, point_3f normal, point_3f color)
{
  point_3f accum = rp.ambient * color;
  for (int i = 0; i < MaxLightsNum; ++i)
  {
    if (!rp.lights[i].enabled)
      continue;

    point_3f lightDir = rp.lights[i].pos - pos;
    float lightDist2 = dot(lightDir, lightDir);
    float lightDist = sqrtf(lightDist2);
    float attenuation = 1.0f / dot(point_3f(1.0f, lightDist, lightDist2), rp.lights[i].attenuationCoefs);
    lightDir /= lightDist;

    point_3f diffuse = rp.lights[i].diffuse * color * max(dot(lightDir, normal), 0.0f);
    
    point_3f viewerDir = normalize(rp.eyePos - pos);
    point_3f hv = normalize(viewerDir + lightDir);
    point_3f specular = rp.lights[i].specular * pow(max(0.0f, dot(hv, normal)), rp.specularExp);

    accum += (diffuse + specular) * attenuation;
  }
  return accum;
}

__device__ point_3f SampleWldPos(int xi, int yi)
{
  int tid = yi * rp.viewWidth + xi;
  return rp.eyePos + rp.rays[tid].t*rp.rays[tid].dir;
}

__device__ point_3f SampleNormal(int xi, int yi)
{
  //int tid = yi * rp.viewWidth + xi;
  int step = 1;
  //int step = max((int)(rp.rays[tid].endNodeSize / (rp.rays[tid].t*rp.detailCoef)), 1);
  
  if (xi < step || xi > rp.viewWidth-step-1)
    return make_float3(0, 0, 1);
  if (yi < step || yi > rp.viewHeight-step-1)
    return make_float3(0, 0, 1);

  point_3f u1 = SampleWldPos(xi-step, yi);
  point_3f u2 = SampleWldPos(xi+step, yi);
  point_3f du = u2-u1;
  point_3f v1 = SampleWldPos(xi, yi-step);
  point_3f v2 = SampleWldPos(xi, yi+step);
  point_3f dv = v2-v1;
  point_3f n = normalize(cross(du, dv));

  return n;
}

__global__ void ShadeSimple(uchar4 * img)
{
  INIT_THREAD

  VoxNodeId node = rp.rays[tid].endNode;
  if (IsNull(node))
  {
    img[tid] = make_uchar4(0, node == EmptyNode ? 0 : 64, 0, 255);
    return;
  }

  float3 p = rp.eyePos;                          
  float3 dir = rp.rays[tid].dir;
  float t = rp.rays[tid].t;

  VoxData vd;
  int childId = rp.rays[tid].endNodeChild;
  if (childId < 0)
    vd = GetVoxData(node);
  else
    vd = GetChild(GetNodePtr(node), childId);

  Color16  c16;
  Normal16 n16;
  UnpackVoxData(vd, c16, n16);
  uchar4 col;
  col = UnpackColorCU(c16);

  point_3f norm;
  //if (((xi/256 + yi/256) & 1) != 0)
    UnpackNormal(n16, norm.x, norm.y, norm.z);
  //else
  //  norm = SampleNormal(xi, yi);

  float3 pt = p + dir*t;
  point_3f materialColor = point_3f(col.x, col.y, col.z) / 256.0f;
  point_3f res = fminf(CalcLighting(pt, norm, materialColor) * 256.0f, point_3f(255, 255, 255));

  img[tid] = make_uchar4(res.x, res.y, res.z, 255);
}

__device__ float4 c2f(const uchar4 & v) { return make_float4(v.x, v.y, v.z, v.w); }

__global__ void Blur(const uchar4 * src, uchar4 * dst)
{
  INIT_THREAD
  
  const int rad = 1;

  if (xi < rad || yi < rad || xi > rp.viewWidth-1-rad || yi > rp.viewHeight-1-rad)
    return;

  float4 accum = make_float4(0);
  for (int y = -rad; y <= rad; ++y)
    for (int x = -rad; x <= rad; ++x)
      accum += c2f(src[(yi+y)*rp.viewWidth + (xi+x)]);
  
  int diam = 2*rad+1;
  accum /= diam*diam;
  dst[tid] = make_uchar4(accum.x, accum.y, accum.z, accum.w);
}

__global__ void BlendLayer(float t1, float t2, const uchar4 * color, uchar4 * dst)
{
  INIT_THREAD;

  float t = rp.rays[tid].t;
  if (t < t1 || t2 <= t)
    return;

  uchar4 s = color[tid];
  uchar4 d = dst[tid];
  
  float da = d.w / 255.0;
  d.x *= da;
  d.y *= da;
  d.z *= da;

  float ds = 1 - da;
  s.x *= ds;
  s.y *= ds;
  s.z *= ds;

  uchar4 res;
  res.x = d.x + s.x; 
  res.y = d.y + s.y; 
  res.z = d.z + s.z; 
  res.w = max(d.w + s.w, 255); 
  dst[tid] = res;
}


extern "C" {

void Run_InitEyeRays(GridShape grid, float * noiseBuf)
{
  InitEyeRays<<<grid.grid, grid.block>>>(noiseBuf);
}

void Run_Trace(GridShape grid)
{
  Trace<<<grid.grid, grid.block>>>();
}

void Run_ShadeSimple(GridShape grid, uchar4 * img)
{
  ShadeSimple<<<grid.grid, grid.block>>>(img);
}

void Run_Blur(GridShape grid, const uchar4 * src, uchar4 * dst)
{
  Blur<<<grid.grid, grid.block>>>(src, dst);
}

void Run_BlendLayer(GridShape grid, float t1, float t2, const uchar4 * color, uchar4 * dst)
{
  BlendLayer<<<grid.grid, grid.block>>>(t1, t2, color, dst);
}

}