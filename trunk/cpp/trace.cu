#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "trace_cu.h"
#include "trace_utils.h"

#define USE_TEXLOOKUP

#define INIT_THREAD \
  const int xi = blockIdx.x * blockDim.x + threadIdx.x; \
  const int yi = blockIdx.y * blockDim.y + threadIdx.y; \
  const int sx = rp.viewWidth;                          \
  const int sy = rp.viewHeight;                         \
  if (xi >= sx || yi >= sy ) return; \
  const int tid = yi*sx + xi;        \

__constant__ VoxStructTree tree;
__constant__ RenderParams rp;


__constant__ float BlurZKernel[BlurZKernSize][BlurZKernSize];
const int halfKernSize = BlurZKernSize / 2;

__constant__ float EdgeThresholdCoef = 4.0f;

texture<uint, 1, hipReadModeElementType> nodes_tex;


#define GET_FIELD( id, fld ) ( tree.nodes[id].fld )

#ifdef USE_TEXLOOKUP
  #define NODE_SZ (sizeof(VoxNode)/4)
  #define GET_TEXNODE_FIELD( p, fld ) ( tex1Dfetch(nodes_tex, (p)+(fld)) )

  typedef uint NodePtr;
  __constant__ NodePtr InvalidPtr = 0xffffffff;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return id*NODE_SZ;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p/NODE_SZ; }

  __device__ VoxNodeInfo GetNodeInfo(NodePtr p) { return GET_TEXNODE_FIELD(p, 0); }
  __device__ VoxNodeId   GetParent  (NodePtr p) { return GET_TEXNODE_FIELD(p, 1); }
  __device__ VoxChild    GetChild   (NodePtr p, int chId) { return GET_TEXNODE_FIELD(p, 3 + chId); }
#else
  typedef const VoxNode * NodePtr;
  __constant__ NodePtr InvalidPtr = NULL;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return tree.nodes + id;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p - tree.nodes; }

  __device__ const VoxNodeInfo & GetNodeInfo(NodePtr p) { return p->flags; }
  __device__ const VoxNodeId   & GetParent  (NodePtr p) { return p->parent; }
  __device__ const VoxChild    & GetChild   (NodePtr p, int chId) { return p->child[chId]; }
#endif

__device__ VoxData GetVoxData  (VoxNodeId id) { return tree.nodes[id].data; }


__device__ float3 CalcRayDirView(int xi, int yi)
{
  const int sx = rp.viewWidth;
  const int sy = rp.viewHeight;
  return point_3f(2*rp.fovCoef*(float)(xi-sx/2)/sx, 2*rp.fovCoef*(float)(yi-sy/2)/sx, -1);
}

__device__ float3 CalcRayDirWorld(int xi, int yi)
{
  point_3f dir = CalcRayDirView(xi, yi);
  dir = rp.viewToWldMtx * dir;
  dir -= rp.eyePos;
  return dir;
}

/*__global__ void InitFishEyeRays(RayData * rays)
{
  INIT_THREAD

  
  float2 p = make_float2(xi-sx/2, yi-sy/2);
  p /= 0.5f*sx;

  const float pi = 3.141593f;
  p *= 0.8;

  float r = length(p);
  float phi = atan2(p.y, p.x);
  float theta = pi/2-asin(r);
  
  float ct = __cosf(theta);
  float3 v = make_float3(__cosf(phi)*ct, __sinf(phi)*ct, __sinf(theta));

  float3 dir = v.x*rp.right + v.y*rp.up + v.z*rp.dir;
  dir = normalize(dir);        

  rays[tid].dir.x = dir.x;
  rays[tid].dir.y = dir.y;
  rays[tid].dir.z = dir.z;

  rays[tid].endNode = 0;
  rays[tid].endNodeChild = EmptyNode;
}*/

__global__ void Trace()
{
  INIT_THREAD

  rp.rays[tid].endNode = 0;
  rp.rays[tid].endNodeChild = EmptyNode;
  rp.zbuf[tid] = 100.0;


  if (IsNull(rp.rays[tid].endNode))
    return;

  point_3f dir = CalcRayDirWorld(xi, yi);
  AdjustDir(dir);

  point_3f t1, t2;
  uint dirFlags = 0;
  if (!SetupTrace(rp.eyePos, dir, t1, t2, dirFlags)) //rp.eyePos
  {
    rp.rays[tid].endNode = EmptyNode;
    return;
  }

  NodePtr nodePtr = GetNodePtr(tree.root);
  int childId = 0;
  int level = 0;
  float nodeSize = pow(0.5f, level);

  enum States { ST_EXIT, ST_ANALYSE, ST_SAVE, ST_GOUP, ST_GODOWN, ST_GONEXT };
  int state = ST_ANALYSE;
  while (state != ST_EXIT)
  {
    switch (state)
    {
      case ST_ANALYSE:
      {
        childId = -1;
        if (maxCoord(t1) * rp.detailCoef > nodeSize/2)  { state = GetEmptyFlag(GetNodeInfo(nodePtr)) ? ST_GOUP : ST_SAVE; break; }
        
        childId = FindFirstChild(t1, t2);
        state = ST_GODOWN;
        break;
      }
      
      case ST_GODOWN:
      {
        if (minCoord(t2) < 0) { state = ST_GONEXT; break; }

        if (GetLeafFlag(GetNodeInfo(nodePtr), childId^dirFlags)) { state = ST_SAVE; break; }
        
        VoxNodeId ch = GetChild(nodePtr, childId^dirFlags);
        if (IsNull(ch)) {state = ST_GONEXT; break; }
        nodePtr = GetNodePtr(ch);
        ++level;
        nodeSize /= 2;
        state = ST_ANALYSE;
        break;
      }
      
      case ST_GONEXT:
      {
        state = GoNext(childId, t1, t2) ? ST_GODOWN : ST_GOUP;
        break;
      }

      case ST_GOUP:
      {
        VoxNodeId p = GetParent(nodePtr);
        if (IsNull(p)) 
        { 
          rp.rays[tid].endNode = EmptyNode;
          state = ST_EXIT; 
          break; 
        }

        for (int i = 0; i < 3; ++i)
        {
          int mask = 1<<i;
          float dt = t2[i] - t1[i];
          ((childId & mask) == 0) ? t2[i] += dt : t1[i] -= dt;
        }
        childId = GetSelfChildId(GetNodeInfo(nodePtr))^dirFlags;
        nodePtr = GetNodePtr(p);
        --level;
        nodeSize *= 2;
        state = ST_GONEXT;
        break;
      }

      case ST_SAVE:
      {
        rp.rays[tid].endNode = Ptr2Id(nodePtr);
        rp.rays[tid].endNodeChild = childId^dirFlags;
        rp.zbuf[tid] = maxCoord(t1);
        rp.rays[tid].endNodeSize = nodeSize;
        state = ST_EXIT;
        break;
      }
    }
  }
}

__device__ point_3f CalcLighting(point_3f pos, point_3f normal, point_3f color)
{
  point_3f accum = rp.ambient * color;
  for (int i = 0; i < MaxLightsNum; ++i)
  {
    if (!rp.lights[i].enabled)
      continue;

    point_3f ligthPos = rp.wldToViewMtx * rp.lights[i].pos;
    point_3f lightDir = ligthPos - pos;
    float lightDist2 = dot(lightDir, lightDir);
    float lightDist = sqrtf(lightDist2);
    float attenuation = 1.0f / dot(point_3f(1.0f, lightDist, lightDist2), rp.lights[i].attenuationCoefs);
    lightDir /= lightDist;

    point_3f diffuse = rp.lights[i].diffuse * color * max(dot(lightDir, normal), 0.0f);
    
    point_3f viewerDir = normalize(point_3f(0, 0, 0) - pos);
    point_3f hv = normalize(viewerDir + lightDir);
    point_3f specular = rp.lights[i].specular * pow(max(0.0f, dot(hv, normal)), rp.specularExp);

    accum += (diffuse + specular) * attenuation;
  }
  return accum;
}

__device__ float absmin(float a, float b)
{
  return abs(a) <  abs(b) ? a : b;
}

__device__ float approxDeriv(float z0, float z1, float z2)
{
  float threshold = EdgeThresholdCoef * z1 * rp.pixelAng;

  float dz = 0;
  float c = 0;
  float d1 = z1 - z0;
  float d2 = z2 - z1;
  if (abs(d1) < threshold)
  {
    dz += d1;
    c += 1.0f;
  }
  if (abs(d2) < threshold)
  {
    dz += d2;
    c += 1.0f;
  }
  if (c == 0)
    return (d1 + d2) / 2;
  return dz / c;
}

__device__ point_3f SampleNormal(int xi, int yi, const float * zbuf)
{
  int tid = yi * rp.viewWidth + xi;
  if (xi == 0 || yi == 0 || xi == rp.viewWidth-1 || yi == rp.viewHeight-1 )
    return point_3f(0, 0, 1);
  
  float z = zbuf[tid];
  float zl = zbuf[tid-1];
  float zr = zbuf[tid+1];
  float zd = zbuf[tid-rp.viewWidth];
  float zu = zbuf[tid+rp.viewWidth];

  float dx = approxDeriv(zl, z, zr);
  float dy = approxDeriv(zd, z, zu);
  
  //nx = -d * dx * z
  //ny = -d * dy * z
  //nz = d * d * z * z
  float d = 2*rp.fovCoef / rp.viewWidth;
  point_3f n(dx, dy, d*z);
  n *= d*z;
  return normalize(n);
}

__global__ void ShadeSimple(uchar4 * img, const float * zbuf )
{
  INIT_THREAD;

  VoxNodeId node = rp.rays[tid].endNode;
  if (IsNull(node))
  {
    img[tid] = make_uchar4(0, node == EmptyNode ? 0 : 64, 0, 255);
    return;
  }

  float3 dir = CalcRayDirView(xi, yi);
  float dl = length(dir);
  dir /= dl;
  float t = zbuf[tid] / dl;

  VoxData vd;
  int childId = rp.rays[tid].endNodeChild;
  if (childId < 0)
    vd = GetVoxData(node);
  else
    vd = GetChild(GetNodePtr(node), childId);

  Color16  c16;
  Normal16 n16;
  UnpackVoxData(vd, c16, n16);
  uchar4 col;
  col = UnpackColorCU(c16);

  point_3f norm;
  if (!rp.ssna)
  {
    UnpackNormal(n16, norm.x, norm.y, norm.z);
    norm = rp.wldToViewMtx * (norm + rp.eyePos);
  }
  else
    norm = SampleNormal(xi, yi, zbuf);


  float3 pt = dir*t;
  point_3f materialColor = point_3f(col.x, col.y, col.z) / 256.0f;
  point_3f res = fminf(CalcLighting(pt, norm, materialColor) * 256.0f, point_3f(255, 255, 255));

  if (rp.showNormals)
    res = norm*255;

  img[tid] = make_uchar4(res.x, res.y, res.z, 255);
}

__global__ void BlurZ(float farLimit, const float * src, float * dst)
{
  INIT_THREAD;
  
  int x1 = max(xi - halfKernSize, 0);
  int x2 = min(xi + halfKernSize, rp.viewWidth-1);
  int y1 = max(yi - halfKernSize, 0);
  int y2 = min(yi + halfKernSize, rp.viewHeight-1);

  int kx = xi - halfKernSize;
  int ky = yi - halfKernSize;

  float z = src[tid];
  if (z > farLimit)
  {
    dst[tid] = z;
    return;
  }
  float threshold = EdgeThresholdCoef * z * rp.pixelAng;
  threshold = max(threshold, 3.0 / 2048);


  float acc = 0, count = 0;
  for (int y = y1; y <= y2; ++y)
  {
    for (int x = x1; x <= x2; ++x)
    {
      float s = src[y * rp.viewWidth + x];
      float dz = s - z;
      if (dz > threshold) // too far
      {
        s = z + 0.3*threshold;
      }
      else if (dz < - threshold) // too close
        s = z;

      float k = BlurZKernel[y - ky][x - kx];
      acc += s * k;
      count += k;
    }
  }
  dst[tid] = acc / count;
}

__global__ void BleedZ(const float * src, float * dst)
{
  INIT_THREAD;

  float z  = src[tid];
  float z0 = z;
  float z1 = z;
  float z2 = z;


  int halfKernSize = 3;

  int x1 = max(xi - halfKernSize, 0);
  int x2 = min(xi + halfKernSize, rp.viewWidth-1);
  int y1 = max(yi - halfKernSize, 0);
  int y2 = min(yi + halfKernSize, rp.viewHeight-1);

  int h2 = halfKernSize*halfKernSize;

  for (int y = y1; y <= y2; ++y)
  {
    for (int x = x1; x <= x2; ++x)
    {
      int dx = x - xi;
      int dy = y - yi;
      if (dx*dx + dy*dy >= h2)
        continue;
      float s = src[y * rp.viewWidth + x];
      if (s < z0)
      {
        z2 = z1;
        z1 = z0;
        z0 = s;
      }
      else if (s < z1)
      {
        z2 = z1;
        z1 = s;
      }
      else if (s < z2)
        z2 = s;
    }
  }

  if (z - z2 > 10.0 / 2048)
    dst[tid] = z2;
  else
    dst[tid] = z;

}


extern "C" {

void Run_Trace(GridShape grid)
{
  Trace<<<grid.grid, grid.block>>>();
}

void Run_ShadeSimple(GridShape grid, uchar4 * img, const float * zbuf)
{
  ShadeSimple<<<grid.grid, grid.block>>>(img, zbuf);
}

void Run_BlurZ(GridShape grid, float farLimit, const float * src, float * dst)
{
  BlurZ<<<grid.grid, grid.block>>>(farLimit, src, dst);
}

void Run_BleedZ(GridShape grid, const float * src, float * dst)
{
  BleedZ<<<grid.grid, grid.block>>>(src, dst);
}


}