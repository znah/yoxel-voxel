#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "trace_cu.h"
#include "trace_utils.h"

#define USE_TEXLOOKUP

#define INIT_THREAD \
  const int xi = blockIdx.x * blockDim.x + threadIdx.x; \
  const int yi = blockIdx.y * blockDim.y + threadIdx.y; \
  const int sx = rp.viewWidth;                          \
  const int sy = rp.viewHeight;                         \
  if (xi >= sx || yi >= sy ) return; \
  const int tid = yi*sx + xi;        \

__constant__ VoxStructTree tree;
__constant__ RenderParams rp;

texture<uint, 1, hipReadModeElementType> nodes_tex;


#define GET_FIELD( id, fld ) ( tree.nodes[id].fld )

#ifdef USE_TEXLOOKUP
  #define NODE_SZ (sizeof(VoxNode)/4)
  #define GET_TEXNODE_FIELD( p, fld ) ( tex1Dfetch(nodes_tex, (p)+(fld)) )

  typedef uint NodePtr;
  __constant__ NodePtr InvalidPtr = 0xffffffff;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return id*NODE_SZ;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p/NODE_SZ; }

  __device__ VoxNodeInfo GetNodeInfo(NodePtr p) { return GET_TEXNODE_FIELD(p, 0); }
  __device__ VoxNodeId   GetParent  (NodePtr p) { return GET_TEXNODE_FIELD(p, 1); }
  __device__ VoxChild    GetChild   (NodePtr p, int chId) { return GET_TEXNODE_FIELD(p, 3 + chId); }
#else
  typedef const VoxNode * NodePtr;
  __constant__ NodePtr InvalidPtr = NULL;

  __device__ NodePtr GetNodePtr(VoxNodeId id) { return tree.nodes + id;  }
  __device__ VoxNodeId Ptr2Id(NodePtr p) { return p - tree.nodes; }

  __device__ const VoxNodeInfo & GetNodeInfo(NodePtr p) { return p->flags; }
  __device__ const VoxNodeId   & GetParent  (NodePtr p) { return p->parent; }
  __device__ const VoxChild    & GetChild   (NodePtr p, int chId) { return p->child[chId]; }
#endif

__device__ VoxData GetVoxData  (VoxNodeId id) { return tree.nodes[id].data; }


__device__ float3 CalcRayDirView(int xi, int yi)
{
  const int sx = rp.viewWidth;
  const int sy = rp.viewHeight;
  return point_3f(2*rp.fovCoef*(float)(xi-sx/2)/sx, 2*rp.fovCoef*(float)(yi-sy/2)/sx, -1);
}

__device__ float3 CalcRayDirWorld(int xi, int yi)
{
  point_3f dir = CalcRayDirView(xi, yi);
  dir = rp.viewToWldMtx * dir;
  dir = dir - rp.eyePos;
  return dir;
}

/*__global__ void InitFishEyeRays(RayData * rays)
{
  INIT_THREAD

  
  float2 p = make_float2(xi-sx/2, yi-sy/2);
  p /= 0.5f*sx;

  const float pi = 3.141593f;
  p *= 0.8;

  float r = length(p);
  float phi = atan2(p.y, p.x);
  float theta = pi/2-asin(r);
  
  float ct = __cosf(theta);
  float3 v = make_float3(__cosf(phi)*ct, __sinf(phi)*ct, __sinf(theta));

  float3 dir = v.x*rp.right + v.y*rp.up + v.z*rp.dir;
  dir = normalize(dir);        

  rays[tid].dir.x = dir.x;
  rays[tid].dir.y = dir.y;
  rays[tid].dir.z = dir.z;

  rays[tid].endNode = 0;
  rays[tid].endNodeChild = EmptyNode;
}*/

__global__ void Trace()
{
  INIT_THREAD

  rp.rays[tid].endNode = 0;
  rp.rays[tid].endNodeChild = EmptyNode;


  if (IsNull(rp.rays[tid].endNode))
    return;

  point_3f dir = CalcRayDirWorld(xi, yi);

  point_3f t1, t2;
  uint dirFlags = 0;
  if (!SetupTrace(rp.eyePos, dir, t1, t2, dirFlags)) //rp.eyePos
  {
    rp.rays[tid].endNode = EmptyNode;
    return;
  }

  NodePtr nodePtr = GetNodePtr(tree.root);
  int childId = 0;
  int level = 0;
  float nodeSize = pow(0.5f, level);

  enum States { ST_EXIT, ST_ANALYSE, ST_SAVE, ST_GOUP, ST_GODOWN, ST_GONEXT };
  int state = ST_ANALYSE;
  while (state != ST_EXIT)
  {
    switch (state)
    {
      case ST_ANALYSE:
      {
        childId = -1;
        if (maxCoord(t1) * rp.detailCoef > nodeSize/2)  { state = GetEmptyFlag(GetNodeInfo(nodePtr)) ? ST_GOUP : ST_SAVE; break; }
        
        childId = FindFirstChild(t1, t2);
        state = ST_GODOWN;
        break;
      }
      
      case ST_GODOWN:
      {
        if (minCoord(t2) < 0) { state = ST_GONEXT; break; }

        if (GetLeafFlag(GetNodeInfo(nodePtr), childId^dirFlags)) { state = ST_SAVE; break; }
        
        VoxNodeId ch = GetChild(nodePtr, childId^dirFlags);
        if (IsNull(ch)) {state = ST_GONEXT; break; }
        nodePtr = GetNodePtr(ch);
        ++level;
        nodeSize /= 2;
        state = ST_ANALYSE;
        break;
      }
      
      case ST_GONEXT:
      {
        state = GoNext(childId, t1, t2) ? ST_GODOWN : ST_GOUP;
        break;
      }

      case ST_GOUP:
      {
        VoxNodeId p = GetParent(nodePtr);
        if (IsNull(p)) 
        { 
          rp.rays[tid].endNode = EmptyNode;
          state = ST_EXIT; 
          break; 
        }

        for (int i = 0; i < 3; ++i)
        {
          int mask = 1<<i;
          float dt = t2[i] - t1[i];
          ((childId & mask) == 0) ? t2[i] += dt : t1[i] -= dt;
        }
        childId = GetSelfChildId(GetNodeInfo(nodePtr))^dirFlags;
        nodePtr = GetNodePtr(p);
        --level;
        nodeSize *= 2;
        state = ST_GONEXT;
        break;
      }

      case ST_SAVE:
      {
        rp.rays[tid].endNode = Ptr2Id(nodePtr);
        rp.rays[tid].endNodeChild = childId^dirFlags;
        rp.zBuf[tid] = maxCoord(t1);
        rp.rays[tid].endNodeSize = nodeSize;
        state = ST_EXIT;
        break;
      }
    }
  }
}

__device__ point_3f CalcLighting(point_3f pos, point_3f normal, point_3f color)
{
  point_3f accum = rp.ambient * color;
  for (int i = 0; i < MaxLightsNum; ++i)
  {
    if (!rp.lights[i].enabled)
      continue;

    point_3f lightDir = rp.lights[i].pos - pos;
    float lightDist2 = dot(lightDir, lightDir);
    float lightDist = sqrtf(lightDist2);
    float attenuation = 1.0f / dot(point_3f(1.0f, lightDist, lightDist2), rp.lights[i].attenuationCoefs);
    lightDir /= lightDist;

    point_3f diffuse = rp.lights[i].diffuse * color * max(dot(lightDir, normal), 0.0f);
    
    point_3f viewerDir = normalize(rp.eyePos - pos);
    point_3f hv = normalize(viewerDir + lightDir);
    point_3f specular = rp.lights[i].specular * pow(max(0.0f, dot(hv, normal)), rp.specularExp);

    accum += (diffuse + specular) * attenuation;
  }
  return accum;
}

/*__device__ point_3f SampleWldPos(int xi, int yi)
{
  int tid = yi * rp.viewWidth + xi;
  return rp.eyePos + rp.rays[tid].t*rp.rays[tid].dir;
}

__device__ point_3f SampleNormal(int xi, int yi)
{
  //int tid = yi * rp.viewWidth + xi;
  int step = 1;
  //int step = max((int)(rp.rays[tid].endNodeSize / (rp.rays[tid].t*rp.detailCoef)), 1);
  
  if (xi < step || xi > rp.viewWidth-step-1)
    return make_float3(0, 0, 1);
  if (yi < step || yi > rp.viewHeight-step-1)
    return make_float3(0, 0, 1);

  point_3f u1 = SampleWldPos(xi-step, yi);
  point_3f u2 = SampleWldPos(xi+step, yi);
  point_3f du = u2-u1;
  point_3f v1 = SampleWldPos(xi, yi-step);
  point_3f v2 = SampleWldPos(xi, yi+step);
  point_3f dv = v2-v1;
  point_3f n = normalize(cross(du, dv));

  return n;
}*/

__global__ void ShadeSimple(uchar4 * img)
{
  INIT_THREAD

  VoxNodeId node = rp.rays[tid].endNode;
  if (IsNull(node))
  {
    img[tid] = make_uchar4(0, node == EmptyNode ? 0 : 64, 0, 255);
    return;
  }

  float3 p = rp.eyePos;                          
  float3 dir = CalcRayDirWorld(xi, yi);
  float dl = length(dir);
  dir /= dl;
  float t = rp.zBuf[tid] / dl;

  VoxData vd;
  int childId = rp.rays[tid].endNodeChild;
  if (childId < 0)
    vd = GetVoxData(node);
  else
    vd = GetChild(GetNodePtr(node), childId);

  Color16  c16;
  Normal16 n16;
  UnpackVoxData(vd, c16, n16);
  uchar4 col;
  col = UnpackColorCU(c16);

  point_3f norm;
  //if (((xi/256 + yi/256) & 1) != 0)
    UnpackNormal(n16, norm.x, norm.y, norm.z);
  //else
  //  norm = SampleNormal(xi, yi);

  float3 pt = p + dir*t;
  point_3f materialColor = point_3f(col.x, col.y, col.z) / 256.0f;
  point_3f res = fminf(CalcLighting(pt, norm, materialColor) * 256.0f, point_3f(255, 255, 255));

  img[tid] = make_uchar4(res.x, res.y, res.z, 255);
}


extern "C" {

void Run_Trace(GridShape grid)
{
  Trace<<<grid.grid, grid.block>>>();
}

void Run_ShadeSimple(GridShape grid, uchar4 * img)
{
  ShadeSimple<<<grid.grid, grid.block>>>(img);
}
}